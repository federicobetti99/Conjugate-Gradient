#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "cg.hh" 
/* -------------------------------------------------------------------------- */
#include <math.h>
#include <iostream>
#include <exception>
/* -------------------------------------------------------------------------- */

const double NEARZERO = 1.0e-14;
const bool DEBUG = true;

__global__ void MatVec(const int N, const int BLOCK_WIDTH, const int BLOCK_HEIGHT, Matrix A, double* p, double* Ap) {
    __shared__ int blockElt;
    __shared__ int blockxInd;
    __shared__ int blockyInd;

    if (threadIdx.x == 0) {
        if ((blockIdx.y + 1) * BLOCK_HEIGHT <= N)
            blockElt = BLOCK_HEIGHT;
        else blockElt = N % BLOCK_HEIGHT;
        blockxInd = blockIdx.x * BLOCK_WIDTH;
        blockyInd = blockIdx.y * BLOCK_HEIGHT;
    }

    __syncthreads();

    // summing variable
    double cSum = 0.;
    int threadxInd = blockxInd + threadIdx.x;

    // make sure we are inside the array horizontally
    if (threadxInd < N) {

        // go through the threads vertically and sum them into a variable
        for (int i = 0; i < blockElt; i++)
            cSum += A(blockyInd + i, threadxInd) * p[blockyInd + i];

        // atomic add these variables to the corresponding c index
        atomicAdd(Ap + threadxInd , cSum);
    }
}

__global__ void MatVec_naive(int N, Matrix A, double* p, double* Ap) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        for (unsigned int j = 0; j < N; ++j) {
            Ap[i] = Ap[i] + A(i, j) * p[j];
        }
    }
}

__global__ void sumVec(int N, double alpha, double* a, double beta, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = alpha * a[i] + beta * b[i];
}

__global__ void fill(int N, double* a, double val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = val;
}

__global__ void copy(int N, double* a, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = b[i];
}

void CGSolver::solve(double* x, const int BLOCK_WIDTH, const int BLOCK_HEIGHT) {
    double *r;
    double *p;
    double *Ap;
    double *tmp;

    hipMallocManaged(&r, m_n * sizeof(double));
    hipMallocManaged(&p, m_n * sizeof(double));
    hipMallocManaged(&Ap, m_n * sizeof(double));
    hipMallocManaged(&tmp, m_n * sizeof(double));

    double conj, rsnew, rsold;
    double *conj_, *rsnew_, *rsold_;
    hipMallocManaged(&conj_, sizeof(double));
    hipMallocManaged(&rsnew_, sizeof(double));
    hipMallocManaged(&rsold_, sizeof(double));

    // define grid size for linear combination of vectors
    dim3 block_size(BLOCK_WIDTH);
    dim3 vec_grid_size((int) ceil(m_n / (double) block_size.x));
    dim3 matvec_grid_size((int) ceil(m_n / (double) BLOCK_WIDTH), (int) ceil(m_m / (double) BLOCK_HEIGHT));
    
    // initialize cublas handle
    hipblasHandle_t h;
    hipblasCreate(&h);

    // initialize vectors
    fill<<<vec_grid_size, block_size>>>(m_n,  x, 0.0);
    fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);

    // r = b - A * x;
    MatVec<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, BLOCK_HEIGHT, m_A, x, Ap);
    hipDeviceSynchronize();
    copy<<<vec_grid_size, block_size>>>(m_n, r, m_b);
    sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -1., Ap);

    // p = r
    copy<<<vec_grid_size, block_size>>>(m_n, p, r);
    
    // rsold = r' * r;
    hipblasDdot(h, m_n, r, 1, p, 1, rsold_);
    hipMemcpy(&rsold, rsold_, sizeof(double), hipMemcpyDeviceToHost);

    // for i = 1:length(b)
    int k = 0;
    for (; k < m_n; ++k) {

        // Ap = A * p;
        fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);
        MatVec<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, BLOCK_HEIGHT, m_A, p, Ap);
        hipDeviceSynchronize();

        // alpha = rsold / (p' * Ap);
        hipblasDdot(h, m_n, p, 1, Ap, 1, conj_);
        hipMemcpy(&conj, conj_, sizeof(double), hipMemcpyDeviceToHost);
        double alpha = rsold / std::max(conj, rsold * NEARZERO);
        
        // x = x + alpha * p;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., x, alpha, p);

        // r = r - alpha * Ap;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -alpha, Ap);

        // rsnew = r' * r;
        hipblasDdot(h, m_n, r, 1, r, 1, rsnew_);
        hipMemcpy(&rsnew, rsnew_, sizeof(double), hipMemcpyDeviceToHost);

        /// CUDA: synchronize to be sure about computation of the residual
        hipDeviceSynchronize();
        
        if (std::sqrt(rsnew) < m_tolerance) break; // Convergence test
            
        // p = r + (rsnew / rsold) * p;
        double beta = rsnew / rsold;
        sumVec<<<vec_grid_size, block_size>>>(m_n, beta, p, 1., r);

        // prepare next iteration and print statistics
        rsold = rsnew;
        if (DEBUG) std::cout << "\t[STEP " << k << "] residual = " << std::scientific << std::sqrt(rsold) << std::endl;
    }

    if (DEBUG) {
        fill<<<vec_grid_size, block_size>>>(m_n, r, 0.0);
        MatVec<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, BLOCK_HEIGHT, m_A, x, r);
        hipDeviceSynchronize();
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -1., m_b);
        double* num_;
        double* denom_;
        hipMallocManaged(&num_, sizeof(double));
        hipMallocManaged(&denom_, sizeof(double));
        double num = 0.;
        double denom = 0.;
        hipblasDdot(h, m_n, r, 1, r, 1, num_);
        hipblasDdot(h, m_n, m_b, 1, m_b, 1, denom_);
        hipMemcpy(&num, num_, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&denom, denom_, sizeof(double), hipMemcpyDeviceToHost);
        auto res = num / denom;
        double* nx_;
        hipMallocManaged(&nx_, sizeof(double));
        double nx = 0.;
        hipblasDdot(h, m_n, x, 1, x, 1, nx_);
        hipMemcpy(&nx, nx_, sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "\t[STEP " << k << "] residual = " << std::scientific
                  << std::sqrt(rsold) << ", ||x|| = " << std::sqrt(nx)
                  << ", ||Ax - b||/||b|| = " << std::sqrt(res) << std::endl;
    }
   
    hipFree(&r);
    hipFree(&tmp);
    hipFree(&p);
    hipFree(&Ap);

    hipblasDestroy(h);
}

void CGSolver::read_matrix(const std::string & filename) {
  m_A.read(filename);
  m_m = m_A.m();
  m_n = m_A.n();
}

/*
Initialization of the source term b
*/
void CGSolver::init_source_term(double h) {
  hipMallocManaged(&m_b, m_n * sizeof(double));
  for (int i = 0; i < m_n; i++) {
    m_b[i] = -2. * i * M_PI * M_PI * std::sin(10. * M_PI * i * h) *
             std::sin(10. * M_PI * i * h);
  }
}
