#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "cg.hh" 
/* -------------------------------------------------------------------------- */
#include <iostream>
#include <exception>
/* -------------------------------------------------------------------------- */

const double NEARZERO = 1.0e-14;
const bool DEBUG = false;
#define PER_ROW

__global__ void MatMulKernel(const int N, const dim3 grid_size, const dim3 block_size, Matrix A, double* p, double* Ap) {
    // get variables for loop
    __shared__ int blockElt;
    __shared__ int blockxInd;
    __shared__ int blockyInd;
    if (threadIdx.x == 0) {
        if ((blockIdx.x + 1) * blockDim.x <= N)
            blockElt = blockDim.x;
        else blockElt = N % blockDim.x;
        blockxInd = blockIdx.x * blockDim.x;
        blockyInd = blockIdx.y * blockDim.y;
    }

    __syncthreads();

    // copy section of b into shared mem
    // use the first BLOCK_WIDTH of thread
    extern __shared__ double b[];

    if (threadIdx.x < blockElt)
        b[threadIdx.x] = p[blockxInd + threadIdx.x];

    __syncthreads();

    // summing variable
    double cSum = 0.;
    int threadyInd = blockyInd + threadIdx.x;

    // make sure we are inside the matrix vertically
    if (threadyInd < N) {

        // go through the threads vertically and sum them into a variable
        for (int i = 0; i < blockElt; i++)
            cSum += b[i] * A((blockxInd + i) * N, threadyInd);

        // atomic add these variables to the corresponding c index
        atomicAdd(Ap + threadyInd, cSum);
    }

}

__global__ void MatVec(int N, Matrix A, double* p, double* Ap) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        for (unsigned int j = 0; j < N; ++j) {
            Ap[i] = Ap[i] + A(i, j) * p[j];
        }
    }
}

__global__ void sumVec(int N, double alpha, double* a, double beta, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = alpha * a[i] + beta * b[i];
}

__global__ void fill(int N, double* a, double val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = val;
}

__global__ void copy(int N, double* a, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = b[i];
}

void CGSolver::kerneled_solve(double* x, dim3 block_size, std::string KERNEL_TYPE) {
    double *r;
    double *p;
    double *Ap;
    double *tmp;

    hipMallocManaged(&r, m_n * sizeof(double));
    hipMallocManaged(&p, m_n * sizeof(double));
    hipMallocManaged(&Ap, m_n * sizeof(double));
    hipMallocManaged(&tmp, m_n * sizeof(double));

    double conj, rsnew, rsold;
    double *conj_, *rsnew_, *rsold_;
    hipMallocManaged(&conj_, sizeof(double));
    hipMallocManaged(&rsnew_, sizeof(double));
    hipMallocManaged(&rsold_, sizeof(double));

    // define grid size for linear combination of vectors
    dim3 vec_grid_size;
    vec_grid_size.x = m_m/block_size.x + (m_m % block_size.x == 0 ? 0 : 1);
    vec_grid_size.y = 1;

    // define grid size for matrix vector products, check on input is done in cg_main.cc
    dim3 matvec_grid_size;
    if (!strcmp(KERNEL_TYPE.c_str(), "NAIVE")) matvec_grid_size = vec_grid_size;
    else {
        matvec_grid_size.x = m_m/block_size.x + (m_m % block_size.x == 0 ? 0 : 1);
        matvec_grid_size.y = m_n/block_size.y + (m_n % block_size.y == 0 ? 0 : 1);
    }
    // initialize cublas handle
    hipblasHandle_t h;
    hipblasCreate(&h);

    // initialize vectors
    fill<<<vec_grid_size, vec_block_size>>>(m_n,  x, 0.0);
    fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);

    // r = b - A * x;
    MatVec<<<matvec_grid_size, block_size>>>(m_n, m_A, x, Ap);
    copy<<<grid_size, block_size>>>(m_n, r, m_b);
    sumVec<<<grid_size, block_size>>>(m_n, 1., r, -1., Ap);

    // p = r
    copy<<<grid_size, block_size>>>(m_n, p, r);
    
    // rsold = r' * r;
    hipblasDdot(h, m_n, r, 1, p, 1, rsold_);
    hipMemcpy(&rsold, rsold_, sizeof(double), hipMemcpyDeviceToHost);

    // for i = 1:length(b)
    int k = 0;
    for (; k < m_n; ++k) {

        // Ap = A * p;
        fill<<<grid_size, block_size>>>(m_n, Ap, 0.0);
        MatVec<<<matvec_grid_size, block_size>>>(m_n, m_A, p, Ap);

        // alpha = rsold / (p' * Ap);
        hipblasDdot(h, m_n, p, 1, Ap, 1, conj_);
        hipMemcpy(&conj, conj_, sizeof(double), hipMemcpyDeviceToHost);
        double alpha = rsold / std::max(conj, rsold * NEARZERO);
        
        // x = x + alpha * p;
        sumVec<<<grid_size, block_size>>>(m_n, 1., x, alpha, p);

        // r = r - alpha * Ap;
        sumVec<<<grid_size, block_size>>>(m_n, 1., r, -alpha, Ap);

        // rsnew = r' * r;
        hipblasDdot(h, m_n, r, 1, r, 1, rsnew_);
        hipMemcpy(&rsnew, rsnew_, sizeof(double), hipMemcpyDeviceToHost);

        // synchronize to be sure about computation of the residual
        hipDeviceSynchronize();
        
        if (std::sqrt(rsnew) < m_tolerance) break; // Convergence test
            
        // p = r + (rsnew / rsold) * p;
        double beta = rsnew / rsold;
        sumVec<<<grid_size, block_size>>>(m_n, beta, p, 1., r);

        // prepare next iteration and print statistics
        rsold = rsnew;
        if (DEBUG) std::cout << "\t[STEP " << k << "] residual = " << std::scientific << std::sqrt(rsold) << std::endl;
    }

    if (DEBUG) {
        fill<<<grid_size, block_size>>>(m_n, r, 0.0);
        MatVec<<<grid_size, block_size>>>(m_n, m_A, x, r);
        sumVec<<<grid_size, block_size>>>(m_n, 1., r, -1., m_b);
        double* num_;
        double* denom_;
        hipMallocManaged(&num_, sizeof(double));
        hipMallocManaged(&denom_, sizeof(double));
        double num = 0.;
        double denom = 0.;
        hipblasDdot(h, m_n, r, 1, r, 1, num_);
        hipblasDdot(h, m_n, m_b, 1, m_b, 1, denom_);
        hipMemcpy(&num, num_, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&denom, denom_, sizeof(double), hipMemcpyDeviceToHost);
        auto res = num / denom;
        double* nx_;
        hipMallocManaged(&nx_, sizeof(double));
        double nx = 0.;
        hipblasDdot(h, m_n, x, 1, x, 1, nx_);
        hipMemcpy(&nx, nx_, sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "\t[STEP " << k << "] residual = " << std::scientific
                  << std::sqrt(rsold) << ", ||x|| = " << nx
                  << ", ||Ax - b||/||b|| = " << res << std::endl;
    }
   
    hipFree(&r);
    hipFree(&tmp);
    hipFree(&p);
    hipFree(&Ap);

    hipblasDestroy(h);
}

void CGSolver::read_matrix(const std::string & filename) {
  m_A.read(filename);
  m_m = m_A.m();
  m_n = m_A.n();
}

/*
Initialization of the source term b
*/
void CGSolver::init_source_term(double h) {
  hipMallocManaged(&m_b, m_n * sizeof(double));
  for (int i = 0; i < m_n; i++) {
    m_b[i] = -2. * i * M_PI * M_PI * std::sin(10. * M_PI * i * h) *
             std::sin(10. * M_PI * i * h);
  }
}
