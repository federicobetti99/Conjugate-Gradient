#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "cg.hh" 
/* -------------------------------------------------------------------------- */
#include <math.h>
#include <iostream>
#include <exception>
/* -------------------------------------------------------------------------- */

const double NEARZERO = 1.0e-14;
const bool DEBUG = true;

__global__ void MatMulKernel(const int N, const int BLOCK_WIDTH, Matrix A, double* p, double* Ap) {
    // get variables for loop
    __shared__ int blockElt;
    if (blockIdx.x * BLOCK_WIDTH + BLOCK_WIDTH <= N) blockElt = BLOCK_WIDTH;
    else blockElt = N % BLOCK_WIDTH;

    __shared__ double b[BLOCK_WIDTH];

    if (threadIdx.x < blockElt)
        b[threadIdx.x] = p[blockIdx.x * BLOCK_WIDTH + threadIdx.x];

    // summing variable
    double cSum = 0.;
    int threadyInd = blockIdx.y * blockDim.x + threadIdx.x;

    // make sure we are inside the matrix vertically
    if (threadyInd < N) {

        // go through the threads vertically and sum them into a variable
        for (int i = 0; i < blockElt; i++)
            cSum += b[i] * A(blockIdx.x * BLOCK_WIDTH + i, threadyInd);

        // atomic add these variables to the corresponding output index
        atomicAdd(Ap + threadyInd, cSum);
    }

}

__global__ void MatVec(int N, Matrix A, double* p, double* Ap) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        for (unsigned int j = 0; j < N; ++j) {
            Ap[i] = Ap[i] + A(i, j) * p[j];
        }
    }
}

__global__ void sumVec(int N, double alpha, double* a, double beta, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = alpha * a[i] + beta * b[i];
}

__global__ void fill(int N, double* a, double val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = val;
}

__global__ void copy(int N, double* a, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = b[i];
}

void CGSolver::solve(double* x, dim3 block_size, int BLOCK_WIDTH) {
    double *r;
    double *p;
    double *Ap;
    double *tmp;

    hipMallocManaged(&r, m_n * sizeof(double));
    hipMallocManaged(&p, m_n * sizeof(double));
    hipMallocManaged(&Ap, m_n * sizeof(double));
    hipMallocManaged(&tmp, m_n * sizeof(double));

    double conj, rsnew, rsold;
    double *conj_, *rsnew_, *rsold_;
    hipMallocManaged(&conj_, sizeof(double));
    hipMallocManaged(&rsnew_, sizeof(double));
    hipMallocManaged(&rsold_, sizeof(double));

    // define grid size for linear combination of vectors
    dim3 vec_grid_size((int) ceil(m_m / (double) block_size.x));
    dim3 matvec_grid_size((int) ceil(m_n / (double) BLOCK_WIDTH), (int) ceil(m_m / (double) block_size.x));
    
    // initialize cublas handle
    hipblasHandle_t h;
    hipblasCreate(&h);

    // initialize vectors
    fill<<<vec_grid_size, block_size>>>(m_n,  x, 0.0);
    fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);

    // r = b - A * x;
    MatMulKernel<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, m_A, x, Ap);
    hipDeviceSynchronize();
    copy<<<vec_grid_size, block_size>>>(m_n, r, m_b);
    sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -1., Ap);

    // p = r
    copy<<<vec_grid_size, block_size>>>(m_n, p, r);
    
    // rsold = r' * r;
    hipblasDdot(h, m_n, r, 1, p, 1, rsold_);
    hipMemcpy(&rsold, rsold_, sizeof(double), hipMemcpyDeviceToHost);

    // for i = 1:length(b)
    int k = 0;
    for (; k < m_n; ++k) {

        // Ap = A * p;
        fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);
        MatMulKernel<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, m_A, p, Ap);
        hipDeviceSynchronize();

        // alpha = rsold / (p' * Ap);
        hipblasDdot(h, m_n, p, 1, Ap, 1, conj_);
        hipMemcpy(&conj, conj_, sizeof(double), hipMemcpyDeviceToHost);
        double alpha = rsold / std::max(conj, rsold * NEARZERO);
        
        // x = x + alpha * p;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., x, alpha, p);

        // r = r - alpha * Ap;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -alpha, Ap);

        // rsnew = r' * r;
        hipblasDdot(h, m_n, r, 1, r, 1, rsnew_);
        hipMemcpy(&rsnew, rsnew_, sizeof(double), hipMemcpyDeviceToHost);

        /// CUDA: synchronize to be sure about computation of the residual
        hipDeviceSynchronize();
        
        if (std::sqrt(rsnew) < m_tolerance) break; // Convergence test
            
        // p = r + (rsnew / rsold) * p;
        double beta = rsnew / rsold;
        sumVec<<<vec_grid_size, block_size>>>(m_n, beta, p, 1., r);

        // prepare next iteration and print statistics
        rsold = rsnew;
        if (DEBUG) std::cout << "\t[STEP " << k << "] residual = " << std::scientific << std::sqrt(rsold) << std::endl;
    }

    if (DEBUG) {
        fill<<<vec_grid_size, block_size>>>(m_n, r, 0.0);
        MatMulKernel<<<matvec_grid_size, block_size>>>(m_n, BLOCK_WIDTH, m_A, x, r);
        hipDeviceSynchronize();
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -1., m_b);
        double* num_;
        double* denom_;
        hipMallocManaged(&num_, sizeof(double));
        hipMallocManaged(&denom_, sizeof(double));
        double num = 0.;
        double denom = 0.;
        hipblasDdot(h, m_n, r, 1, r, 1, num_);
        hipblasDdot(h, m_n, m_b, 1, m_b, 1, denom_);
        hipMemcpy(&num, num_, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&denom, denom_, sizeof(double), hipMemcpyDeviceToHost);
        auto res = num / denom;
        double* nx_;
        hipMallocManaged(&nx_, sizeof(double));
        double nx = 0.;
        hipblasDdot(h, m_n, x, 1, x, 1, nx_);
        hipMemcpy(&nx, nx_, sizeof(double), hipMemcpyDeviceToHost);
        std::cout << "\t[STEP " << k << "] residual = " << std::scientific
                  << std::sqrt(rsold) << ", ||x|| = " << std::sqrt(nx)
                  << ", ||Ax - b||/||b|| = " << std::sqrt(res) << std::endl;
    }
   
    hipFree(&r);
    hipFree(&tmp);
    hipFree(&p);
    hipFree(&Ap);

    hipblasDestroy(h);
}

void CGSolver::read_matrix(const std::string & filename) {
  m_A.read(filename);
  m_m = m_A.m();
  m_n = m_A.n();
}

/*
Initialization of the source term b
*/
void CGSolver::init_source_term(double h) {
  hipMallocManaged(&m_b, m_n * sizeof(double));
  for (int i = 0; i < m_n; i++) {
    m_b[i] = -2. * i * M_PI * M_PI * std::sin(10. * M_PI * i * h) *
             std::sin(10. * M_PI * i * h);
  }
}
