#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "cg.hh" 
/* -------------------------------------------------------------------------- */
#include <math.h>
#include <iostream>
#include <exception>
#include <map>
#include <cstring>
/* -------------------------------------------------------------------------- */

const double NEARZERO = 1.0e-14;
const bool DEBUG = true;

__global__ void MatVec(const int N, const int NUM_THREADS, const int BLOCK_WIDTH,
                        Matrix A, double* p, double* Ap)
{

    /**
    * Efficient kernel for matrix vector product, every thread takes care of the dot product between a subpart of a
    * row of A and the corresponding subpart of p, then atomicAdd from the same thread in every block is done.
    * Coalesced memory accesses are not favoured by using symmetry of A, unlike the kernel MatVecT. In particular,
    * every thread takes care of BLOCK_WIDTH elements of a row of A, and threads with the same idx in the block
    * are collaborating to the computation of the corresponding entry of Ap.
    *
    * @param N Size of the matrix (always assumed square)
    * @param BLOCK_WIDTH width of the block
    * @param BLOCK_HEIGHT height of the block
    * @param A matrix
    * @param p vector
    * @param Ap vector for the result of A*p
    * @return void
    */

    __shared__ int blockElt;
    __shared__ int blockxInd;
    __shared__ int blockyInd;

    if ((blockIdx.x + 1) * BLOCK_WIDTH <= N)
        blockElt = BLOCK_WIDTH;
    else blockElt = N % BLOCK_WIDTH;
    blockxInd = blockIdx.x * BLOCK_WIDTH;
    blockyInd = blockIdx.y * NUM_THREADS;

    // summing variable
    double cSum = 0.;
    int threadyInd = blockyInd + threadIdx.x;

    // make sure we are inside the array horizontally
    if (threadyInd < N) {

        // go through the threads vertically and sum them into a variable
        for (int i = 0; i < blockElt; i++)
            cSum += A(threadyInd, blockxInd + i) * p[blockxInd + i];

        atomicAdd(Ap + threadyInd, cSum);
    }

}

__global__ void MatVecT(const int N, const int NUM_THREADS, const int BLOCK_WIDTH,
                                Matrix A, double* p, double* Ap)
{

    /**
    * Efficient kernel for matrix vector product, every thread takes care of the dot product between a subpart of a
    * row of A and the corresponding subpart of p, then atomicAdd from the same thread in every block is done.
    * Coalesced memory accesses are favoured by exploiting symmetry of A. In particular, every thread takes care of
    * BLOCK_WIDTH elements of a row of A, and threads with the same idx in the block are collaborating to the
    * computation of the corresponding entry of Ap. This kernel provided overall the best results.
    *
    * @param N Size of the matrix (always assumed square)
    * @param BLOCK_WIDTH width of the block
    * @param BLOCK_HEIGHT height of the block
    * @param A matrix
    * @param p vector
    * @param Ap vector for the result of A*p
    * @return void
    */

    // define common variables to all the elements of the block
    __shared__ int blockElt;
    __shared__ int blockxInd;
    __shared__ int blockyInd;

    if ((blockIdx.y + 1) * BLOCK_WIDTH <= N)
        blockElt = BLOCK_WIDTH;
    else blockElt = N % BLOCK_WIDTH;
    blockxInd = blockIdx.x * NUM_THREADS;
    blockyInd = blockIdx.y * BLOCK_WIDTH;

    // summing variable
    double cSum = 0.;
    int threadxInd = blockxInd + threadIdx.x;

    // make sure we are inside the array horizontally
    if (threadxInd < N) {

        // go through the threads vertically and sum them into a variable
        for (int i = 0; i < blockElt; i++)
            cSum += A(blockyInd + i, threadxInd) * p[blockyInd + i];

        atomicAdd(Ap + threadxInd, cSum);
    }

}

__global__ void sumVec(int N, double alpha, double* a, double beta, double* b)
{

    /**
    * Simple kernel for summing of two vectors, here the optimization of the topology behind leaves space to much
    * less details, every thread takes care of summing one element of the two vectors
    *
    * @param N Size of the vectors a and b
    * @param alpha coefficient to multiply a
    * @param a vector to be summed premultiplied by alpha
    * @param beta coefficient to multiply b
    * @param b vector to be summed premultiplied by beta
    * @return void
    */

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = alpha * a[i] + beta * b[i];

}

__global__ void fill(int N, double* a, double val)
{

    /**
    * Simple kernel to set all the elements of the vector a to value val
    *
    * @param N Size of the vector a
    * @param a vector to be filled
    * @param val value to fill in all the elements of a
    * @return void
    */

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = val;

}

__global__ void copy(int N, double* a, double* b)
{

    /**
    * Simple kernel to copy the content of vector b into the content of vector a
    *
    * @param N Size of the vector a
    * @param a vector to be filled
    * @param b vector to be copied into a
    * @return void
    */

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) a[i] = b[i];

}

void CGSolver::solve(double* x, const int NUM_THREADS, const int BLOCK_WIDTH, const bool T)
{

    /**
    * Main function to solve the linear system Ax=b with conjugate gradient
    *
    * @param x initial guess, zero vector usually
    * @param BLOCK_WIDTH width of the block for CUDA kernels
    * @param NUM_THREADS number of threads per block
    * @param T true to use transposed kernel for matrix vector products, thus favouring coalesced memory access
    * @return void
    */

    double *r;
    double *p;
    double *Ap;
    double *tmp;

    hipMallocManaged(&r, m_n * sizeof(double));
    hipMallocManaged(&p, m_n * sizeof(double));
    hipMallocManaged(&Ap, m_n * sizeof(double));
    hipMallocManaged(&tmp, m_n * sizeof(double));

    double conj, rsnew, rsold;
    double *conj_, *rsnew_, *rsold_;
    hipMallocManaged(&conj_, sizeof(double));
    hipMallocManaged(&rsnew_, sizeof(double));
    hipMallocManaged(&rsold_, sizeof(double));

    // define grid size for linear combination of vectors
    dim3 block_size(NUM_THREADS);
    dim3 vec_grid_size(ceil(m_n / (double) NUM_THREADS));

    // grid size for matrix vector products
    dim3 matvec_grid_size;
    if (T) {
        // blocks are arranged vertically exploiting symmetry of A
        matvec_grid_size.x = ceil(m_n / (double) NUM_THREADS);
        matvec_grid_size.y = ceil(m_m / (double) BLOCK_WIDTH);
    }
    else {
        // blocks are arranged horizontally, not exploiting symmetry of A
        matvec_grid_size.x = ceil(m_n / (double) BLOCK_WIDTH);
        matvec_grid_size.y = ceil(m_m / (double) NUM_THREADS);
    }

    // initialize cublas handle
    hipblasHandle_t h;
    hipblasCreate(&h);

    // initialize vectors
    fill<<<vec_grid_size, block_size>>>(m_n,  x, 0.0);
    fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);

    if (T) MatVecT<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, x, Ap);
    else MatVec<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, x, Ap);
    hipDeviceSynchronize();

    copy<<<vec_grid_size, block_size>>>(m_n, r, m_b);
    sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -1., Ap);

    // p = r
    copy<<<vec_grid_size, block_size>>>(m_n, p, r);
    
    // rsold = r' * r;
    hipblasDdot(h, m_n, r, 1, p, 1, rsold_);
    hipMemcpy(&rsold, rsold_, sizeof(double), hipMemcpyDeviceToHost);

    // for i = 1:length(b)
    int k = 0;
    for (; k < m_n; ++k) {

        // Ap = A * p;
        fill<<<vec_grid_size, block_size>>>(m_n, Ap, 0.0);
        if (T) MatVecT<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, p, Ap);
        else MatVec<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, p, Ap);
        hipDeviceSynchronize();  // synchronize as topology changes between matrix vector products and other operations

        // alpha = rsold / (p' * Ap);
        hipblasDdot(h, m_n, p, 1, Ap, 1, conj_);
        hipMemcpy(&conj, conj_, sizeof(double), hipMemcpyDeviceToHost);
        double alpha = rsold / std::max(conj, rsold * NEARZERO);
        
        // x = x + alpha * p;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., x, alpha, p);

        // r = r - alpha * Ap;
        sumVec<<<vec_grid_size, block_size>>>(m_n, 1., r, -alpha, Ap);

        // rsnew = r' * r;
        hipblasDdot(h, m_n, r, 1, r, 1, rsnew_);
        hipMemcpy(&rsnew, rsnew_, sizeof(double), hipMemcpyDeviceToHost);

        /// CUDA: synchronize to be sure about computation of the residual
        hipDeviceSynchronize();
        
        if (std::sqrt(rsnew) < m_tolerance) break; // Convergence test
            
        // p = r + (rsnew / rsold) * p;
        double beta = rsnew / rsold;
        sumVec<<<vec_grid_size, block_size>>>(m_n, beta, p, 1., r);

        // prepare next iteration and print statistics
        rsold = rsnew;
    }

    if (DEBUG) {
       double* r;
       double nx;
       double* nx_;
       double nb;
       double* nb_;
       hipMallocManaged(&nb_, sizeof(double));
       hipMallocManaged(&nx_, sizeof(double));
       hipMallocManaged(&r, m_n * sizeof(double));
       fill<<<vec_grid_size, block_size>>>(m_n, r, 0.0);  
       if (T) MatVecT<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, x, Ap);
       else MatVec<<<matvec_grid_size, block_size>>>(m_n, NUM_THREADS, BLOCK_WIDTH, m_A, x, Ap);
       hipDeviceSynchronize();
       copy<<<vec_grid_size, block_size>>>(m_n, r, m_b);
       sumVec<<<vec_grid_size, block_size>>>(m_n, 1.0, r, -1.0, Ap);
       hipblasDdot(h, m_n, r, 1, r, 1, rsnew_);
       hipMemcpy(&rsnew, rsnew_, sizeof(double), hipMemcpyDeviceToHost);
       hipblasDdot(h, m_n, x, 1, x, 1, nx_);
       hipMemcpy(&nx, nx_, sizeof(double), hipMemcpyDeviceToHost);
       hipblasDdot(h, m_n, m_b, 1, m_b, 1, nb_);
       hipMemcpy(&nb, nb_, sizeof(double), hipMemcpyDeviceToHost);         
       std::cout << "\t[STEP " << k << "] residual = " << std::scientific
              << std::sqrt(rsold) << ", ||x|| = " << std::sqrt(nx)
              << ", ||Ax - b||/||b|| = " << std::sqrt(rsnew) / std::sqrt(nb) << std::endl;
    }

    hipFree(&r);
    hipFree(&tmp);
    hipFree(&p);
    hipFree(&Ap);

    hipblasDestroy(h);

}

void CGSolver::read_matrix(const std::string & filename)
{

    /**
    * Read matrix from file and set problem size
    *
    * @param filename filename
    * @return void
    */

    m_A.read(filename);
    m_m = m_A.m();
    m_n = m_A.n();

}


void CGSolver::init_source_term(double h)
{

    /**
    * Initialization of source term
    *
    * @param h step size
    * @return void
    */

    hipMallocManaged(&m_b, m_n * sizeof(double));
    for (int i = 0; i < m_n; i++) {
    m_b[i] = -2. * i * M_PI * M_PI * std::sin(10. * M_PI * i * h) *
             std::sin(10. * M_PI * i * h);
    }

}
