#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "cg.hh" 
/* -------------------------------------------------------------------------- */
#include <iostream>
#include <exception>
/* -------------------------------------------------------------------------- */

const double NEARZERO = 1.0e-14;

__global__ void matrix_vector_product(double* A, double* p, double* Ap, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double tmp = 0.;
    for (unsigned int j = 0; j < N; ++j) {
        tmp += A[i * N + j] * p[j];
    }
    Ap[i] = tmp;
}

__global__ void vector_sum(double* a, double alpha, double* b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] += alpha * b[i];
}

__global__ void scalar_product(double* a, double* b, double* result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    result[i] = a[i] * b[i];
}

void CGSolver::kerneled_solve(double* x, dim3 block_size) {
    double *r, *p, *Ap, *tmp;
    hipMallocManaged(&r, m_n * sizeof(double));
    hipMallocManaged(&p, m_n * sizeof(double));
    hipMallocManaged(&Ap, m_n * sizeof(double));
    hipMallocManaged(&tmp, m_n * sizeof(double));
 
    for (int i = 0; i < m_n; i++) Ap[i] = 0.;
    for (int i = 0; i < m_n; i++) tmp[i] = 0.;

    dim3 grid_size;
    grid_size.x = m_m/block_size.x;
    grid_size.y = 1;

    double* conj;
    hipMallocManaged(&conj, m_n * sizeof(double));
    for (int i = 0; i < m_n; i++) conj[i] = 0.;

    double* rsnew;
    hipMallocManaged(&rsnew, m_n * sizeof(double));
    for (int i = 0; i < m_n; i++) rsnew[i] = 0.;

    // r = b - A * x;
    matrix_vector_product<<<grid_size, block_size>>>(m_A.data(), x, Ap, m_n);
    hipDeviceSynchronize();

    r = m_b;
    vector_sum<<<grid_size, block_size>>>(r, -1., Ap);
    hipDeviceSynchronize();

    // p = r
    p = r;
    
    // rsold = r' * r;
    double* rsold;
    hipMallocManaged(&rsold, m_n * sizeof(double));
    for (int i = 0; i < m_n; i++) rsold[i] = 0.;

    scalar_product<<<grid_size, block_size>>>(r, p, rsold);
    hipDeviceSynchronize();
    for (int i = 1; i < m_n; i++) *rsold += rsold[i];

    // for i = 1:length(b)
    int k = 0;
    for (; k < m_n; ++k) {

        // Ap = A * p
        matrix_vector_product<<<grid_size, block_size>>>(m_A.data(), p, Ap, m_n); 
        hipDeviceSynchronize();        

        // alpha = rsold / (p' * Ap);
        scalar_product<<<grid_size, block_size>>>(p, Ap, conj);
        hipDeviceSynchronize();
        for (int i = 1; i < m_n; i++) *conj += conj[i];
        double alpha = *rsold / std::max(*conj, *rsold * NEARZERO);
        
        // x = x + alpha * p;
        vector_sum<<<grid_size, block_size>>>(x, alpha, p);
        // r = r - alpha * Ap;
        vector_sum<<<grid_size, block_size>>>(r, -alpha, Ap);
        hipDeviceSynchronize();

        // rsnew = r' * r;
        scalar_product<<<grid_size, block_size>>>(r, r, rsnew);
        hipDeviceSynchronize();
        for (int i = 1; i < m_n; i++) *rsnew += rsnew[i]; 

        if (std::sqrt(*rsnew) < m_tolerance) break; // Convergence test
            
        double beta = *rsnew / *rsold;
        // p = r + (rsnew / rsold) * p
        tmp = r;
        vector_sum<<<grid_size, block_size>>>(tmp, beta, p);
        hipDeviceSynchronize();
        p = tmp;

        *rsold = *rsnew;
        std::cout << "\t[STEP " << k << "] residual = " << std::scientific << std::sqrt(*rsold) << std::endl;
    }
   
    hipFree(&r);
    hipFree(&tmp);
    hipFree(&p);
    hipFree(&Ap);
}

void CGSolver::read_matrix(const std::string & filename) {
  m_A.read(filename);
  m_m = m_A.m();
  m_n = m_A.n();
}

/*
Initialization of the source term b
*/
void CGSolver::init_source_term(double h) {
  hipMallocManaged(&m_b, m_n*sizeof(double));
  for (int i = 0; i < m_n; i++) {
    m_b[i] = -2. * i * M_PI * M_PI * std::sin(10. * M_PI * i * h) *
             std::sin(10. * M_PI * i * h);
  }
}
